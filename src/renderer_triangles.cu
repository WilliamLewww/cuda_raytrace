#include "hip/hip_runtime.h"
#include <algorithm>
#include <fstream>
#include <stdio.h>

#include <GL/glew.h>
#include <GLFW/glfw3.h>

#include "structures.h"
#include "model.h"
#include "analysis.h"

#define FRAME_WIDTH 1000
#define FRAME_HEIGHT 1000

#define IMAGE_WIDTH 5000
#define IMAGE_HEIGHT 5000

#define LIGHT_COUNT 1

#define MESH_DESCRIPTOR_COUNT 2
#define MESH_SEGMENT_COUNT 24

#define REFLECTIVE_RAY_EPILSON 0.0001
#define SHADOW_EPILSON 0.00001
#define TRIANGLE_INTERSECTION_EPILSON 0.0000001

__constant__ Camera camera[1];

__constant__ Light lightArray[LIGHT_COUNT];

__constant__ MeshDescriptor meshDescriptorArray[MESH_DESCRIPTOR_COUNT];
__constant__ MeshSegment meshSegmentArray[MESH_SEGMENT_COUNT];

__device__ Tuple operator*(float* matrix, Tuple tuple) { return {(matrix[0] * tuple.x) + (matrix[1] * tuple.y) + (matrix[2] * tuple.z) + (matrix[3] * tuple.w), (matrix[4] * tuple.x) + (matrix[5] * tuple.y) + (matrix[6] * tuple.z) + (matrix[7] * tuple.w), (matrix[8] * tuple.x) + (matrix[9] * tuple.y) + (matrix[10] * tuple.z) + (matrix[11] * tuple.w), (matrix[12] * tuple.x) + (matrix[13] * tuple.y) + (matrix[14] * tuple.z) + (matrix[15] * tuple.w)}; }
__device__ Ray transform(Ray ray, float* matrix) { return {(matrix * ray.origin), (matrix * ray.direction)}; }
__device__ Tuple operator+(Tuple tupleA, Tuple tupleB) { return {tupleA.x + tupleB.x, tupleA.y + tupleB.y, tupleA.z + tupleB.z, tupleA.w + tupleB.w}; }
__device__ Tuple operator-(Tuple tupleA, Tuple tupleB) { return {tupleA.x - tupleB.x, tupleA.y - tupleB.y, tupleA.z - tupleB.z, tupleA.w - tupleB.w}; }
__device__ Tuple operator*(Tuple tuple, float scalar) { return {tuple.x * scalar, tuple.y * scalar, tuple.z * scalar, tuple.w * scalar}; }
__device__ Tuple operator*(float scalar, Tuple tuple) { return {tuple.x * scalar, tuple.y * scalar, tuple.z * scalar, tuple.w * scalar}; }
__device__ Tuple hadamardProduct(Tuple tupleA, Tuple tupleB) { return {tupleA.x * tupleB.x, tupleA.y * tupleB.y, tupleA.z * tupleB.z, tupleA.w * tupleB.w}; }
__device__ float magnitude(Tuple tuple) { return sqrt(tuple.x * tuple.x + tuple.y * tuple.y + tuple.z * tuple.z + tuple.w * tuple.w); }
__device__ Tuple normalize(Tuple tuple) { return {tuple.x / magnitude(tuple), tuple.y / magnitude(tuple), tuple.z / magnitude(tuple), tuple.w / magnitude(tuple)}; }
__device__ Tuple negate(Tuple tuple) { return {-tuple.x, -tuple.y, -tuple.z, -tuple.w}; }
__device__ Tuple project(Ray ray, float t) { return ray.origin + (ray.direction * t); }
__device__ float dot(Tuple tupleA, Tuple tupleB) { return (tupleA.x * tupleB.x) + (tupleA.y * tupleB.y) + (tupleA.z * tupleB.z) + (tupleA.w * tupleB.w); }
__device__ Tuple cross(Tuple tupleA, Tuple tupleB) { return {(tupleA.y * tupleB.z) - (tupleA.z * tupleB.y), (tupleA.z * tupleB.x) - (tupleA.x * tupleB.z), (tupleA.x * tupleB.y) - (tupleA.y * tupleB.x), 1.0f}; }
__device__ Tuple reflect(Tuple tuple, Tuple normal) { return tuple - (normal * 2.0f * dot(tuple, normal)); }

__device__
int intersectTriangle(float* intersectionMagnitude, MeshDescriptor meshDescriptor, MeshSegment meshSegment, Ray ray) {
  Ray transformedRay = transform(ray, meshDescriptor.inverseModelMatrix);

  Tuple edgeB = meshSegment.vertexB - meshSegment.vertexA;
  Tuple edgeC = meshSegment.vertexC - meshSegment.vertexA;

  Tuple h = cross(transformedRay.direction, edgeC);
  float a = dot(edgeB, h);
  float f = 1.0f / a;
  Tuple s = transformedRay.origin - meshSegment.vertexA;
  float u = f * dot(s, h);
  Tuple q = cross(s, edgeB);
  float v = f * dot(transformedRay.direction, q);
  float t = f * dot(edgeC, q);

  int intersecting = (t > TRIANGLE_INTERSECTION_EPILSON && t < 1.0f / TRIANGLE_INTERSECTION_EPILSON) * (a <= -TRIANGLE_INTERSECTION_EPILSON || a >= TRIANGLE_INTERSECTION_EPILSON) * (u >= 0.0f && u <= 1.0f) * (v >= 0.0f && u + v <= 1.0f);
  *intersectionMagnitude = t;

  return intersecting;
}

__device__
Tuple colorFromRay(Ray ray) {
  Tuple color = {0.0f, 0.0f, 0.0f, 0.0f};
  int intersectionIndex = -1;
  int intersectionDescriptorIndex = -1;
  float intersectionMagnitude = 0.0f;

  int segmentOffset = 0;
  #pragma unroll
  for (int y = 0; y < MESH_DESCRIPTOR_COUNT; y++) {
    for (int x = segmentOffset; x < segmentOffset + meshDescriptorArray[y].segmentCount; x++) {
      float point;
      int count = intersectTriangle(&point, meshDescriptorArray[y], meshSegmentArray[x], ray);

      intersectionIndex = (x * (count > 0 && (point < intersectionMagnitude || intersectionMagnitude == 0))) + (intersectionIndex * (count <= 0 || (point >= intersectionMagnitude && intersectionMagnitude != 0)));
      intersectionDescriptorIndex = (y * (count > 0 && (point < intersectionMagnitude || intersectionMagnitude == 0))) + (intersectionDescriptorIndex * (count <= 0 || (point >= intersectionMagnitude && intersectionMagnitude != 0)));
      intersectionMagnitude = (point * (count > 0 && (point < intersectionMagnitude || intersectionMagnitude == 0))) + (intersectionMagnitude * (count <= 0 || (point >= intersectionMagnitude && intersectionMagnitude != 0)));
    }
  }

  if (intersectionIndex != -1) {
    Ray transformedRay = transform(ray, meshDescriptorArray[intersectionDescriptorIndex].inverseModelMatrix);
    Tuple intersectionPoint = project(transformedRay, intersectionMagnitude - SHADOW_EPILSON);
    Ray lightRay = {meshDescriptorArray[intersectionDescriptorIndex].modelMatrix * intersectionPoint, normalize(lightArray[0].position - (meshDescriptorArray[intersectionDescriptorIndex].modelMatrix * intersectionPoint))};

    int intersecionCount = 0;

    segmentOffset = 0;
    #pragma unroll
    for (int y = 0; y < MESH_DESCRIPTOR_COUNT; y++) {
      for (int x = segmentOffset; x < segmentOffset + meshDescriptorArray[y].segmentCount; x++) {
        float point = 0;
        intersecionCount += intersectTriangle(&point, meshDescriptorArray[y], meshSegmentArray[x], lightRay) * (point < magnitude(lightArray[0].position - intersectionPoint));
      }
    }

    float lightNormalDifference = dot(meshSegmentArray[intersectionIndex].normal, lightRay.direction);

    color = (0.1f * meshSegmentArray[intersectionIndex].color) + 
            (0.7f * lightNormalDifference * meshSegmentArray[intersectionIndex].color * (lightNormalDifference > 0) * (intersecionCount == 0));
  }

  return color;
}

__device__
Ray rayFromReflection(Ray ray, int recursionCount = 0) {
  return ray;
}

__global__
void lighting(Tuple* colorOut, int renderWidth, int renderHeight) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (idx >= renderWidth || idy >= renderHeight) { return; }

  Tuple pixel = {
    (idx - (renderWidth / 2.0f)) / renderWidth, 
    (idy - (renderHeight / 2.0f)) / renderHeight, 
    0.0f, 1.0f
  };
  Tuple direction = normalize((pixel + camera[0].direction) - camera[0].position);
  Ray ray = {camera[0].position, direction};
  ray = transform(ray, camera[0].modelMatrix);

  colorOut[(idy*renderWidth)+idx] = colorFromRay(ray);
}

__global__
void reflections(Tuple* colorOut, int renderWidth, int renderHeight) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (idx >= renderWidth || idy >= renderHeight) { return; }

  Tuple pixel = {
    (idx - (renderWidth / 2.0f)) / renderWidth, 
    (idy - (renderHeight / 2.0f)) / renderHeight, 
    0.0f, 1.0f
  };
  Tuple direction = normalize((pixel + camera[0].direction) - camera[0].position);
  Ray ray = {camera[0].position, direction};
  ray = transform(ray, camera[0].modelMatrix);

  colorOut[(idy*renderWidth)+idx] = colorFromRay(rayFromReflection(ray));
}

void writeColorDataToFile(const char* filename, unsigned int* colorData) {
  std::ofstream file;
  file.open(filename);
  file << "P3\n" << IMAGE_WIDTH << " " << IMAGE_HEIGHT << "\n255\n";

  for (int x = 0; x < IMAGE_WIDTH * IMAGE_HEIGHT; x++) {
    file << int(colorData[x] & 0x0000FF) << " ";
    file << int((colorData[x] & 0x00FF00) >> 8) << " ";
    file << int((colorData[x] & 0xFF0000) >> 16) << "\n";
  }

  file.close();
}

__global__
void combineLightingReflectionBuffers(unsigned int* cudaBuffer, Tuple* lightingBuffer, Tuple* reflectionsBuffer, int renderWidth, int renderHeight) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (idx >= renderWidth || idy >= renderHeight) { return; }

  Tuple color;
  if (reflectionsBuffer[(idy*renderWidth)+idx].w > 0) {
    color = (0.2 * reflectionsBuffer[(idy*renderWidth)+idx]) + lightingBuffer[(idy*renderWidth)+idx];
  }
  else {
    color = lightingBuffer[(idy*renderWidth)+idx];
  }

  cudaBuffer[(idy*renderWidth)+idx] = (int(fmaxf(0, fminf(255, color.z))) << 16) | (int(fmaxf(0, fminf(255, color.y))) << 8) | (int(fmaxf(0, fminf(255, color.x))));
}

Tuple* lightingBuffer;
Tuple* reflectionsBuffer;
extern "C" void updateCamera(float x, float y, float z, float rotationX, float rotationY) {
  Camera h_camera[] = {{{0.0, 0.0, 0.0, 1.0}, {0.0, 0.0, 1.0, 0.0}}};
  initializeModelMatrix(h_camera[0].modelMatrix, multiply(multiply(createTranslateMatrix(x, y, z), createRotationMatrixY(rotationY)), createRotationMatrixX(rotationX)));
  initializeInverseModelMatrix(h_camera[0].inverseModelMatrix, h_camera[0].modelMatrix);
  hipMemcpyToSymbol(HIP_SYMBOL(camera), h_camera, sizeof(Camera));
}

extern "C" void initializeScene() {
  hipMalloc(&lightingBuffer, FRAME_WIDTH*FRAME_HEIGHT*sizeof(Tuple));
  hipMalloc(&reflectionsBuffer, FRAME_WIDTH*FRAME_HEIGHT*sizeof(Tuple));

  Camera h_camera[] = {{{0.0, 0.0, 0.0, 1.0}, {0.0, 0.0, 1.0, 0.0}}};
  initializeModelMatrix(h_camera[0].modelMatrix, multiply(multiply(createTranslateMatrix(5.0, -3.5, -6.0), createRotationMatrixY(-M_PI / 4.5)), createRotationMatrixX(-M_PI / 12.0)));
  initializeInverseModelMatrix(h_camera[0].inverseModelMatrix, h_camera[0].modelMatrix);
  hipMemcpyToSymbol(HIP_SYMBOL(camera), h_camera, sizeof(Camera));

  Light h_lightArray[] = {{{10.0, -10.0, -5.0, 1.0}, {1.0, 1.0, 1.0, 1.0}}};
  hipMemcpyToSymbol(HIP_SYMBOL(lightArray), h_lightArray, LIGHT_COUNT*sizeof(Light));

  MeshDescriptor* h_meshDescriptorArray = new MeshDescriptor[MESH_DESCRIPTOR_COUNT];
  MeshSegment* h_meshSegmentArray = new MeshSegment[MESH_SEGMENT_COUNT];

  Model modelA = createModelFromOBJ("res/cube.obj");
  Model modelB = createModelFromOBJ("res/cube.obj");
  initializeModelMatrix(&modelA.meshDescriptor, createScaleMatrix(5.0, 0.15, 5.0));
  initializeModelMatrix(&modelB.meshDescriptor, createTranslateMatrix(0.0, -2.0, 0.0));

  h_meshDescriptorArray[0] = modelA.meshDescriptor;
  h_meshDescriptorArray[1] = modelB.meshDescriptor;
  memcpy(&h_meshSegmentArray[0], modelA.meshSegmentArray, 12*sizeof(MeshSegment));
  memcpy(&h_meshSegmentArray[12], modelB.meshSegmentArray, 12*sizeof(MeshSegment));

  hipMemcpyToSymbol(HIP_SYMBOL(meshDescriptorArray), h_meshDescriptorArray, MESH_DESCRIPTOR_COUNT*sizeof(MeshDescriptor));
  hipMemcpyToSymbol(HIP_SYMBOL(meshSegmentArray), h_meshSegmentArray, MESH_SEGMENT_COUNT*sizeof(MeshSegment));
}

extern "C" void updateScene() {

}

extern "C" void renderFrame(int blockDimX, int blockDimY, void* cudaBuffer, hipGraphicsResource_t* cudaTextureResource) {
  dim3 block(blockDimX, blockDimY);
  dim3 grid((FRAME_WIDTH + block.x - 1) / block.x, (FRAME_HEIGHT + block.y - 1) / block.y);
  lighting<<<grid, block>>>(lightingBuffer, FRAME_WIDTH, FRAME_HEIGHT);
  reflections<<<grid, block>>>(reflectionsBuffer, FRAME_WIDTH, FRAME_HEIGHT);
  combineLightingReflectionBuffers<<<grid, block>>>((unsigned int*)cudaBuffer, lightingBuffer, reflectionsBuffer, FRAME_WIDTH, FRAME_HEIGHT);

  hipArray *texture_ptr;
  hipGraphicsMapResources(1, cudaTextureResource, 0);
  hipGraphicsSubResourceGetMappedArray(&texture_ptr, *cudaTextureResource, 0, 0);

  hipMemcpy2DToArray(texture_ptr, 0, 0,  cudaBuffer, FRAME_WIDTH*4*sizeof(GLubyte), FRAME_WIDTH*4*sizeof(GLubyte), FRAME_HEIGHT, hipMemcpyDeviceToDevice);
  hipGraphicsUnmapResources(1, cudaTextureResource, 0);
}

extern "C" void renderImage(int blockDimX, int blockDimY, const char* filename) {
  printf("\n");

  Analysis::setAbsoluteStart();
  Analysis::createLabel(0, "allocate_memory");
  Analysis::createLabel(1, "execute_kernel");
  Analysis::createLabel(2, "copy_device");
  Analysis::createLabel(3, "create_image");

  Analysis::begin();
  unsigned int* h_imageData = (unsigned int*)malloc(IMAGE_WIDTH*IMAGE_HEIGHT*4*sizeof(GLubyte));
  unsigned int* d_imageData;
  hipMalloc((unsigned int**)&d_imageData, IMAGE_WIDTH*IMAGE_HEIGHT*4*sizeof(GLubyte));

  Tuple *d_lightingData, *d_reflectionsData;
  hipMalloc((Tuple**)&d_lightingData, IMAGE_WIDTH*IMAGE_HEIGHT*sizeof(Tuple));
  hipMalloc((Tuple**)&d_reflectionsData, IMAGE_WIDTH*IMAGE_HEIGHT*sizeof(Tuple));
  Analysis::end(0);

  dim3 block(blockDimX, blockDimY);
  dim3 grid((IMAGE_WIDTH + block.x - 1) / block.x, (IMAGE_HEIGHT + block.y - 1) / block.y);

  Analysis::begin();
  printf("rendering ray traced image...\n");
  lighting<<<grid, block>>>(d_lightingData, IMAGE_WIDTH, IMAGE_HEIGHT);
  reflections<<<grid, block>>>(d_reflectionsData, IMAGE_WIDTH, IMAGE_HEIGHT);
  combineLightingReflectionBuffers<<<grid, block>>>(d_imageData, d_lightingData, d_reflectionsData, IMAGE_WIDTH, IMAGE_HEIGHT);
  hipDeviceSynchronize();
  printf("finished rendering\n");
  Analysis::end(1);

  Analysis::begin();
  hipMemcpy(h_imageData, d_imageData, IMAGE_WIDTH*IMAGE_HEIGHT*4*sizeof(GLubyte), hipMemcpyDeviceToHost);
  Analysis::end(2);

  Analysis::begin();
  writeColorDataToFile(filename, h_imageData);
  printf("saved image as: [%s]\n", filename);
  Analysis::end(3);

  Analysis::printAll(IMAGE_WIDTH, IMAGE_HEIGHT);

  free(h_imageData);
  hipFree(d_imageData);
  hipFree(d_lightingData);
  hipFree(d_reflectionsData);
}
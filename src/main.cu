#include "hip/hip_runtime.h"
#include <fstream>
#include <stdio.h>
#include "analysis.h"
#include "structures.h"

#define IMAGE_WIDTH 1000
#define IMAGE_HEIGHT 1000
#define FOV 1.0471975512

#define SHADE_OFFSET 0.01
#define PLANE_COMPARISON 0.00001

#define LIGHT_COUNT 1

#define SPHERE_COUNT 2
#define PLANE_COUNT 1

__constant__ Camera camera[1];

__constant__ Light lightArray[LIGHT_COUNT];

__constant__ Sphere sphereArray[SPHERE_COUNT];
__constant__ Plane planeArray[PLANE_COUNT];

__device__
int intersectSphere(float* intersectionPoint, Sphere sphere, Ray ray) {
	Tuple sphereToRay = ray.origin - sphere.origin;
	float a = dot(ray.direction, ray.direction);
	float b = 2.0f * dot(sphereToRay, ray.direction);
	float c = dot(sphereToRay, sphereToRay) - 1.0f;

	float discriminant = (b * b) - (4.0f * a * c);
	float pointA = (-b - sqrt(discriminant)) / (2.0f * a);
	float pointB = (-b + sqrt(discriminant)) / (2.0f * a);

	*intersectionPoint = (pointB * (pointA <= pointB)) + (pointA * (pointB < pointA));

	return (discriminant >= 0) * (2 - (pointA == pointB));
}

__device__
int intersectPlane(float* intersectionPoint, Plane plane, Ray ray) {
	int intersectionCount = 0;

	float denom = dot(plane.normal, ray.direction);
	if (fabsf(denom) > PLANE_COMPARISON) {
		float t = dot(plane.origin - ray.origin, plane.normal);

		if (t >= 0) {
			intersectionCount = 1;
			*intersectionPoint = t;
		}
	}

	return intersectionCount;
}

__global__
void colorFromRay(Tuple* colorOut) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (idx >= IMAGE_WIDTH || idy >= IMAGE_HEIGHT) { return; }

	Tuple origin = {0.0f, 0.0f, 0.0f, 1.0f};
	Tuple pixel = {(float(idx) * camera[0].pixelSize) - (camera[0].pixelSize * IMAGE_WIDTH / 2.0f), 
				   (float(idy) * camera[0].pixelSize) - (camera[0].pixelSize * IMAGE_HEIGHT / 2.0f), 100.0f, 1.0f};
	Tuple direction = normalize(pixel - origin);

	Ray ray = {origin, direction};

	int intersectionIndex = -1;
	float intersectionPoint = 0.0f;

	#pragma unroll
	for (int x = 0; x < SPHERE_COUNT; x++) {
		float point;
		int count = intersectSphere(&point, sphereArray[x], ray);

		intersectionIndex = (x * (count > 0 && (point < intersectionPoint || intersectionPoint == 0))) + (intersectionIndex * (count <= 0 || (point >= intersectionPoint && intersectionPoint != 0)));
		intersectionPoint = (point * (count > 0 && (point < intersectionPoint || intersectionPoint == 0))) + (intersectionPoint * (count <= 0 || (point >= intersectionPoint && intersectionPoint != 0)));
	}

	if (intersectionIndex != -1) {
		Tuple intersectionPosition = project(ray, intersectionPoint);
		Ray lightRay = {intersectionPosition, normalize(lightArray[0].position - intersectionPosition)};

		float lightIntersectionPoint = 0.0f;

		#pragma unroll
		for (int x = 0; x < SPHERE_COUNT; x++) {
			float point;
			int count = intersectSphere(&point, sphereArray[x], lightRay);

			lightIntersectionPoint = (point * (count > 0 && (point < lightIntersectionPoint || lightIntersectionPoint == 0))) + (lightIntersectionPoint * (count <= 0 || (point >= lightIntersectionPoint && lightIntersectionPoint != 0)));
		}

		Tuple projectedPosition = project(lightRay, lightIntersectionPoint);
		int inShade = (magnitude(projectedPosition) + SHADE_OFFSET) < magnitude(intersectionPosition);

		Tuple direction = normalize(lightArray[0].position - project(ray, intersectionPoint));
		Tuple normal = normalize(sphereArray[intersectionIndex].origin - project(ray, intersectionPoint));
		float angleDifference = dot(normal, direction);
		float color = (0.1f * 255.0f) + ((angleDifference > 0) * angleDifference) * 255.0f * inShade;

		colorOut[(idy*IMAGE_WIDTH)+idx] = {color, 0.0f, 0.0f, 1.0f};
	}
	else {
		colorOut[(idy*IMAGE_WIDTH)+idx] = {0.0f, 0.0f, 0.0f, 1.0f};
	}
}

void writeColorDataToFile(const char* filename, Tuple* colorData) {
	std::ofstream file;
	file.open(filename);
	file << "P3\n" << IMAGE_WIDTH << " " << IMAGE_HEIGHT << "\n255\n";

	for (int x = 0; x < IMAGE_WIDTH * IMAGE_HEIGHT; x++) {
		file << int(colorData[x].x) << " ";
		file << int(colorData[x].y) << " ";
		file << int(colorData[x].z) << "\n";
	}

	file.close();
}

int main(int argn, char** argv) {
	printf("\n");

	Analysis::setAbsoluteStart();
	Analysis::createLabel(0, "allocate_memory");
	Analysis::createLabel(1, "execute_kernel");
	Analysis::createLabel(2, "copy_device");
	Analysis::createLabel(3, "create_image");

	Analysis::begin();
	float halfView = tan(FOV / 2);
	float aspect = float(IMAGE_WIDTH) / float(IMAGE_HEIGHT);

	float halfWidth = ((aspect >= 1) * halfView) + ((aspect < 1) * halfView * aspect);
	float halfHeight = ((aspect >= 1) * halfView / aspect) + ((aspect < 1) * halfView);
	float pixelSize = ((halfWidth * 2) / IMAGE_WIDTH) * 500;

	const Camera h_camera[] = {{halfWidth, halfHeight, pixelSize}};
	hipMemcpyToSymbol(HIP_SYMBOL(camera), h_camera, sizeof(Camera));

	const Light h_lightArray[] = {{{-10, -10, 5, 1}, {1, 1, 1, 1}}};
	hipMemcpyToSymbol(HIP_SYMBOL(lightArray), h_lightArray, LIGHT_COUNT*sizeof(Light));

	const Sphere h_sphereArray[] = {{{0.0, 0.0, 5.0, 1.0}},{{2.0, 2.0, 5.0, 1.0}}};
	hipMemcpyToSymbol(HIP_SYMBOL(sphereArray), h_sphereArray, SPHERE_COUNT*sizeof(Sphere));

	const Plane h_planeArray[] = {{{0, 0, 5},{0, 1, 1}}};
	hipMemcpyToSymbol(HIP_SYMBOL(planeArray), h_planeArray, PLANE_COUNT*sizeof(Plane));

	Tuple* h_colorData = (Tuple*)malloc(IMAGE_WIDTH*IMAGE_HEIGHT*sizeof(Tuple));
	Tuple* d_colorData;
	hipMalloc((Tuple**)&d_colorData, IMAGE_WIDTH*IMAGE_HEIGHT*sizeof(Tuple));
	Analysis::end(0);

	dim3 block(32, 32);
	dim3 grid((IMAGE_WIDTH + block.x - 1) / block.x, (IMAGE_HEIGHT + block.y - 1) / block.y);

	Analysis::begin();
	printf("rendering ray traced image...\n");
	colorFromRay<<<grid, block>>>(d_colorData);
	hipDeviceSynchronize();
	printf("finished rendering\n");
	Analysis::end(1);

	Analysis::begin();
	hipMemcpy(h_colorData, d_colorData, IMAGE_WIDTH*IMAGE_HEIGHT*sizeof(Tuple), hipMemcpyDeviceToHost);
	hipFree(d_colorData);
	Analysis::end(2);

	Analysis::begin();
	const char* filename = argv[1];
	writeColorDataToFile(filename, h_colorData);
	printf("saved image as: [%s]\n", filename);
	Analysis::end(3);

	Analysis::printAll(IMAGE_WIDTH, IMAGE_HEIGHT);

	hipDeviceReset();
	free(h_colorData);
	printf("\n");
	return 0;
}
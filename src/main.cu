#include "hip/hip_runtime.h"
#include <fstream>
#include <stdio.h>
#include "structures.h"

#define IMAGE_WIDTH 1000
#define IMAGE_HEIGHT 1000

#define LIGHT_COUNT 1
#define SPHERE_COUNT 2

__constant__ Light lightArray[LIGHT_COUNT];
__constant__ Sphere sphereArray[SPHERE_COUNT];

__device__
int intersectSphere(float* intersectionPoint, Sphere sphere, Ray ray) {
	Tuple sphereToRay = ray.origin - sphere.origin;
	float a = dot(ray.direction, ray.direction);
	float b = 2.0f * dot(sphereToRay, ray.direction);
	float c = dot(sphereToRay, sphereToRay) - (sphere.radius * sphere.radius);

	float discriminant = (b * b) - (4.0f * a * c);
	float pointA = (-b - sqrt(discriminant)) / (2.0f * a);
	float pointB = (-b + sqrt(discriminant)) / (2.0f * a);

	*intersectionPoint = (pointA * (pointA <= pointB)) + (pointB * (pointB < pointA));

	return (discriminant >= 0) * (2 - (pointA == pointB)) * (pointA > 0 && pointB > 0);
}

__global__
void colorFromRay(Tuple* colorOut) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (idx >= IMAGE_WIDTH || idy >= IMAGE_HEIGHT) { return; }

	Tuple origin = {0.0f, 0.0f, 0.0f, 1.0f};
	Tuple pixel = {idx - (IMAGE_WIDTH / 2.0f), idy - (IMAGE_HEIGHT / 2.0f), 200.0f, 1.0f};
	Tuple direction = normalize(pixel - origin);

	Ray ray = {origin, direction};

	int intersectionIndex = -1;
	float intersectionPoint = 0.0f;

	#pragma unroll
	for (int x = 0; x < SPHERE_COUNT; x++) {
		float point;
		int count = intersectSphere(&point, sphereArray[x], ray);

		intersectionIndex = (x * (count > 0 && (point < intersectionPoint || intersectionPoint == 0))) + (intersectionIndex * (count <= 0 || (point >= intersectionPoint && intersectionPoint != 0)));
		intersectionPoint = (point * (count > 0 && (point < intersectionPoint || intersectionPoint == 0))) + (intersectionPoint * (count <= 0 || (point >= intersectionPoint && intersectionPoint != 0)));
	}

	if (intersectionIndex != -1) {
		Tuple intersectionPosition = project(ray, intersectionPoint);
		Ray lightRay = {intersectionPosition, normalize(lightArray[0].position - intersectionPosition)};

		int intersecionCount = 0;

		#pragma unroll
		for (int x = 0; x < SPHERE_COUNT; x++) {
			if (x != intersectionIndex) {
				float point;
				intersecionCount += intersectSphere(&point, sphereArray[x], lightRay);
			}
		}

		Tuple direction = normalize(lightArray[0].position - project(ray, intersectionPoint));
		Tuple normal = negate(normalize(sphereArray[intersectionIndex].origin - project(ray, intersectionPoint)));
		float angleDifference = dot(normal, direction);
		Tuple color = (0.1f * sphereArray[intersectionIndex].color) + ((angleDifference > 0) * angleDifference) * sphereArray[intersectionIndex].color * (intersecionCount == 0);

		colorOut[(idy*IMAGE_WIDTH)+idx] = color;
	}
	else {
		colorOut[(idy*IMAGE_WIDTH)+idx] = {0.0, 0.0, 0.0, 1.0};
	}
}

void writeColorDataToFile(const char* filename, Tuple* colorData) {
	std::ofstream file;
	file.open(filename);
	file << "P3\n" << IMAGE_WIDTH << " " << IMAGE_HEIGHT << "\n255\n";

	for (int x = 0; x < IMAGE_WIDTH * IMAGE_HEIGHT; x++) {
		file << int(colorData[x].x) << " ";
		file << int(colorData[x].y) << " ";
		file << int(colorData[x].z) << "\n";
	}

	file.close();
}

int main(int argn, char** argv) {
	printf("\n");

	Tuple* h_colorData = (Tuple*)malloc(IMAGE_WIDTH*IMAGE_HEIGHT*sizeof(Tuple));
	Tuple* d_colorData;
	hipMalloc((Tuple**)&d_colorData, IMAGE_WIDTH*IMAGE_HEIGHT*sizeof(Tuple));

	const Light h_lightArray[] = {{{-10.0, -10.0, 0.0, 1.0}, {1.0, 1.0, 1.0, 1.0}}};
	hipMemcpyToSymbol(HIP_SYMBOL(lightArray), h_lightArray, LIGHT_COUNT*sizeof(Light));

	const Sphere h_sphereArray[] = {
									{{0.0, 0.0, 3.0, 1.0}, 2.0, {255.0, 0.0, 0.0}},
									{{5.0, 5.0, 5.0, 1.0}, 4.0, {0.0, 255.0, 0.0}}
								};
	hipMemcpyToSymbol(HIP_SYMBOL(sphereArray), h_sphereArray, SPHERE_COUNT*sizeof(Sphere));

	dim3 block(32, 32);
	dim3 grid((IMAGE_WIDTH + block.x - 1) / block.x, (IMAGE_HEIGHT + block.y - 1) / block.y);

	colorFromRay<<<grid, block>>>(d_colorData);
	hipDeviceSynchronize();

	hipMemcpy(h_colorData, d_colorData, IMAGE_WIDTH*IMAGE_HEIGHT*sizeof(Tuple), hipMemcpyDeviceToHost);
	hipFree(d_colorData);

	const char* filename = argv[1];
	writeColorDataToFile(filename, h_colorData);
	printf("saved image as: [%s]\n", filename);

	hipDeviceReset();
	free(h_colorData);
	printf("\n");
	return 0;
}